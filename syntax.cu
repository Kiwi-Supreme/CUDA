int main() {
    // Host arrays and size
    int size = 1024;
    int *h_a, *h_b, *h_c;
    h_a = new int[size];
    h_b = new int[size];
    h_c = new int[size];

    // Initialize host arrays h_a and h_b

    // Device arrays
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy host data to device
    hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_b, size);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
