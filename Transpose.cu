#include "hip/hip_runtime.h"
#include <stdio.h>

// Define matrix dimensions
#define N 10


// CUDA kernel to transpose matrices
__global__ void matrixAdd(int *a, int *c)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        c[row * N + col] = a[col * N + row];
    }
}

int main()
{
    int a[N][M], c[N][M];
    int *d_a, *d_c;
    int size = N * N * sizeof(int);

    // Initialize matrices a and b
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            a[i][j] = i + j;
            b[i][j] = i - j;
        }
    }

    // Allocate memory on the GPU
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_c, size);

    // Copy matrices from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid(ceil((float)N / 16), ceil((float)M / 16), 1);
    dim3 dimBlock(16, 16, 1);

    // Launch the kernel
    matrixAdd<<<dimGrid, dimBlock>>>(d_a, d_c);

    // Copy the result matrix from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result matrix
    printf("Matrix C:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    // Free allocated memory on the GPU
    hipFree(d_a);
    hipFree(d_c);

    return 0;
}

