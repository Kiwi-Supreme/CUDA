#include "hip/hip_runtime.h"
#include <stdio.h>


// CUDA kernel to add numbers
__global__ void matrixAdd(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main()
{
    int a=5, b=7, c;
    int *d_a, *d_b, *d_c;
    int size = N * M * sizeof(int);
    

    // Allocate memory on the GPU
    hipMalloc((void *)&d_a, size);
    hipMalloc((void *)&d_b, size);
    hipMalloc((void *)&d_c, size);

    // Copy matrices from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


    // Launch the kernel
    matrixAdd<<<1, 1>>>(d_a, d_b, d_c);

    // Copy the result matrix from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    printf("%d ", c);
    
    // Free allocated memory on the GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

